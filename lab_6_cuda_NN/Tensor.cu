#include "Tensor.h"

Tensor* Tensor::copy2gpu()
{
    Tensor *output = new Tensor();
    hipMalloc(&(output->tensor), this->height * this->width * sizeof(dtype));
    hipMemcpy(output->tensor, this->tensor, this->height * this->width * sizeof(dtype), hipMemcpyHostToDevice);
    output->height = this->height;
    output->width = this->width;
    return output;
}

Tensor* Tensor::copy2cpu()
{
    Tensor *output = new Tensor();
    output->tensor = (dtype*)malloc(this->height * this->width * sizeof(dtype));
    hipMemcpy(output->tensor, this->tensor, this->height * this->width * sizeof(dtype), hipMemcpyDeviceToHost);
    output->height = this->height;
    output->width = this->width;
    return output;
}

Tensor* Tensor::gpu2gpu()
{
    Tensor *output = new Tensor();
    hipMalloc(&(output->tensor), this->height * this->width * sizeof(dtype));
    hipMemcpy(output->tensor, this->tensor, this->height * this->width * sizeof(dtype), hipMemcpyDeviceToDevice);
    output->height = this->height;
    output->width = this->width;
    return output;
}