#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define BLOCK_DIM 16
#define BLOCK_VEC_DIM 256

#define CUDACHKERR(err) if (err != hipSuccess) { \
    fprintf(stderr, \
            "Failed to copy vector B from host to device (error code %s)!\n", \
            hipGetErrorString(err)); \
    exit(EXIT_FAILURE); \
    }

void print_help()
{
    printf("usage:\n");
    printf("{min_error} {matrix_size} {iter_max}\n");
}

double* getSetMatrix(double* dst, int size, hipStream_t stream)
{
    hipError_t err;

    double *matrix;
    err = hipMalloc(&matrix, size * size * sizeof(double));
    CUDACHKERR(err);

    err = hipMemcpyAsync(matrix, dst, size * size * sizeof(double), hipMemcpyHostToDevice, stream);
    CUDACHKERR(err);

    return matrix;
}

void interpolationMatrixSides(double* matrix, int matrix_size)
{
    // left side
    for (int i = 1; i < matrix_size - 1; ++i)
    {
        matrix[i * matrix_size] = matrix[0] * (matrix_size - 1 - i) / (matrix_size - 1) +
                                     matrix[matrix_size * (matrix_size - 1)] * i / (matrix_size - 1);
    }

    // top side
    for (int i = 1; i < matrix_size - 1; ++i)
    {
        matrix[i] = matrix[0] * (matrix_size - 1 - i) / (matrix_size - 1) +
                    matrix[matrix_size - 1] * i / (matrix_size - 1);
    }

    // right side
    for (int i = 1; i < matrix_size - 1; ++i)
    {
        matrix[i * matrix_size + matrix_size - 1] = matrix[matrix_size - 1] * (matrix_size - 1 - i) / (matrix_size - 1) +
                                                        matrix[(matrix_size - 1) * matrix_size + matrix_size - 1] * i / (matrix_size - 1);
    }

    // bottom side
    for (int i = 1; i < matrix_size - 1; ++i)
    {
        matrix[(matrix_size - 1) * matrix_size + i] = matrix[(matrix_size - 1) * matrix_size] * (matrix_size - 1 - i) / (matrix_size - 1) +
                                                       matrix[(matrix_size - 1) * matrix_size + matrix_size - 1] * i / (matrix_size - 1);
    }
}

__global__ void vecNeg(const double *newA, const double *A, double* ans, int numElements)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numElements)
    {
        ans[idx] =  newA[idx] - A[idx];
    }
}

__global__ void evalEquation(double *newA, const double *A, int numElements)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if ((0 < idx && idx < numElements - 1) && (0 < idy && idy < numElements - 1))
    {
        newA[idy * numElements + idx] = 0.25 * (__ldg(&A[(idy - 1) * numElements + idx]) + __ldg(&A[(idy + 1) * numElements + idx]) +
                                                __ldg(&A[idy * numElements + (idx - 1)]) + __ldg(&A[idy * numElements + (idx + 1)]));
    }
}

void printCudaMatrix(double* dst, int size)
{
    double *a = (double*)calloc(sizeof(double), size * size);

    hipMemcpy(a, dst, size * size * sizeof(double), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; ++i)
    {
        for (int j = 0; j < size; ++j)
        {
            printf("%lf ", a[i * size + j]);
        }
        printf("\n");
    }
    printf("\n");

    free(a);
}

void checkCudaInfo()
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties( &prop, 0);

    printf("major = %d \n", prop.major);
    printf("warp Size = %d \n", prop.warpSize);
    printf("max Threads Per Block = %d \n", prop.maxThreadsPerBlock);
    printf("max Threads Per MultiProcessor = %d \n", prop.maxThreadsPerMultiProcessor);
    printf("multiProcessor Count = %d \n", prop.multiProcessorCount);
    printf("shared Memory Per Block (bytes) = %lu \n", prop.sharedMemPerBlock);
    printf("max Grid Size by X = %d \n", prop.maxGridSize[0]);
    printf("max Grid Size by Y = %d \n", prop.maxGridSize[1]);

    printf("\n");
}

int main(int argc, char *argv[])
{
    if (argc == 1)
    {
        checkCudaInfo();
        print_help();
        exit(0);
    }

    double min_error = atof(argv[1]);
    int matrix_size = atoi(argv[2]);
    int iter_max = atoi(argv[3]);

    hipError_t err;
    hipStream_t stream;
    hipStreamCreate(&stream);

    double *tmp = (double*)calloc(sizeof(double), matrix_size * matrix_size);

    tmp[0] = 10.0;
    tmp[matrix_size - 1] = 20.0;
    tmp[(matrix_size - 1) * matrix_size] = 20.0;
    tmp[(matrix_size - 1) * matrix_size + matrix_size - 1] = 30.0;

    interpolationMatrixSides(tmp, matrix_size);

    double *A_d = getSetMatrix(tmp, matrix_size, stream);
    double *newA_d = getSetMatrix(tmp, matrix_size, stream);
    free(tmp);

    int iter = 0;
    double error = 10;

    dim3 BS = dim3(BLOCK_DIM, BLOCK_DIM);

    dim3 GS = dim3(ceil(matrix_size / (double)BS.x), ceil(matrix_size / (double)BS.y));

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    double *tmp_d, *max_d;
    hipMalloc(&tmp_d, sizeof(double) * matrix_size * matrix_size);
    hipMalloc(&max_d, sizeof(double));

    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, tmp_d, max_d, matrix_size * matrix_size);
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    int BS_neg = BLOCK_VEC_DIM;
    int GS_neg = ceil(matrix_size * matrix_size / (double)BS_neg);

    bool graphCreated = false;
    hipGraph_t graph;
    hipGraphExec_t instance;

    hipStreamSynchronize(stream);

    while (error > min_error && iter < iter_max)
    {
        if (!graphCreated)
        {
            hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

            for (int i = 0; i < 100; ++i)
            {
                evalEquation<<<GS, BS, 0, stream>>>(newA_d, A_d, matrix_size);
                tmp = A_d;
                A_d = newA_d;
                newA_d = tmp;
            }
            hipStreamEndCapture(stream, &graph);
            hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
            graphCreated = true;
        }
        err = hipGraphLaunch(instance, stream);
        CUDACHKERR(err);
        err = hipStreamSynchronize(stream);
        CUDACHKERR(err);

        iter += 100;

        if (iter % 100 == 0)
        {
            printf("iter = %d error = %e\n", iter, error);
            error = 0;

            tmp = A_d;
            A_d = newA_d;
            newA_d = tmp;

            vecNeg<<<GS_neg, BS_neg, 0, stream>>>(newA_d, A_d, tmp_d, matrix_size * matrix_size);

            err = hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, tmp_d, max_d, matrix_size * matrix_size, stream);
            CUDACHKERR(err);

            err = hipMemcpyAsync(&error, max_d, sizeof(double), hipMemcpyDeviceToHost, stream);
            CUDACHKERR(err);

            tmp = A_d;
            A_d = newA_d;
            newA_d = tmp;
        }
    }

    hipFree(A_d);
    hipFree(newA_d);
    hipFree(tmp_d);
    hipFree(max_d);
    hipFree(d_temp_storage);

    return 0;
}
